#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdint.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define BATCH_SIZE 4096
#define WARPS_NB 10
#define abs8(n) ((n) & 0x7fu)
#define abs32(n) ((n) & 0x7fffffffu)

struct clause {
	/* Field 'flags':
	 *   0x01u - value of literal l[0] (taking into account the sign)
	 *   0x02u - value of literal l[1] (taking into account the sign)
	 *   0x04u - value of literal l[2] (taking into account the sign)
	 *   0x08u - literal l[0] was assigned a value
	 *   0x10u - literal l[1] was assigned a value
	 *   0x20u - literal l[2] was assigned a value
	 * Satisfied if: (flags & 0x07u) != 0x00u
	 * Has literals: (flags & 0x38u) != 0x38u
	 * Invalid if:   (flags & 0x3fu) == 0x2au
	 */
	int8_t l[3];
	uint8_t flags;
#define c_sat(c) (((c).flags & 0x07u) != 0x00u)
#define c_has(c) (((c).flags & 0x38u) != 0x38u)
#define c_inv(c) (((c).flags & 0x3fu) == 0x38u)
};

/************************* PREPROCESS ****************************/

__global__ void preprocess(clause *d_f1, unsigned int *d_v, int r) {
	int warp_id = WARPS_NB * blockIdx.x + (threadIdx.x >> 5); // check
	int lane_id = threadIdx.x & 31;
	clause *formula = d_f1 + warp_id * r;
	unsigned int *valid = d_v + warp_id; // check

	// dodac ifa jezeli jestesmy warpem niezerowym? on juz ma te dane przeciez?
	// o nie... musi byc osobna tablica... co jak warp 0 juz zabierze sie do roboty?
	for(int i = threadIdx.x & 31; i < r; ++i) {
		formula[i] = d_f1[i];
	}
	
	int number = warp_id;

	while(number) { // check
		int tmp = number / 3;
		int branch_id = number - 3 * tmp;
		number = tmp;
		clause fc;
		bool fc_found = false;
		unsigned int mask1 = 0xffffffffu; // check

		for(int i = lane_id; true; i += 32) {
			mask1 = __ballot_sync(mask1, i < r); // check for second loop

			if(i >= r) {
				break;
			}

			clause cl = formula[i];

			if(!fc_found) {
				int has_literals = c_has(cl); // check and/or improve
				int mask2 = __ballot_sync(mask1, has_literals); // check if it is OK

				if(!mask2) {
					continue;
				}

				fc_found = true;
				int *ptr_cl = (int *) &cl;
				int src_lane_id = __ffs(mask2) - 1;
				tmp = __shfl_sync(mask1, *ptr_cl, src_lane_id);
				fc = *((clause *) &tmp);

				if(!(fc.flags & (0x08u << branch_id))) {
					if(lane_id == 0) {
						*valid = 0;
					}

					return;
				}
			}

			for(int l = 0; l < 3; ++l) {
				for(int x = 0; x < branch_id; ++x) {
					if(!(cl.flags & (0x08u << l)) && abs8(cl.l[l]) == abs8(fc.l[x])) {
						cl.flags |= (0x08u + (fc.l[x] < 0)) << l;
					}
				}

				if(cl.l[l] == fc.l[branch_id]) {
					cl.flags |= (0x08u + (fc.l[branch_id] > 0)) << l;
				}
			}

			if(__any_sync(0xffffffffu, c_inv(cl))) {
				if(lane_id == 0) {
					*valid = 0;
				}

				return;
			}

			formula[i] = cl;
		}

		if(!fc_found) {
			// whole formula satisfied! I think...
			return;
		}
	}
}

/************************* SAT_KERNEL ****************************/

/* Triples the number of formulas in a batch and marks invalid/missing ones
 * 
 * d_f - array of formulas and a free space for new formulas
 * d_v - array of flags indicating whether a formula is valid or not
 * k - number of formulas to triple
 * r - total number of clauses
 */
__global__ void sat_kernel(clause *d_f1, clause *d_f2, unsigned int *d_v, int k, int r) {
	int lane_id = threadIdx.x & 31;
	int warp_id = WARPS_NB * blockIdx.x + (threadIdx.x >> 5);
	int formula_id = warp_id / 3;
	int branch_id = warp_id - 3 * formula_id;
	unsigned int *valid = d_v + k * branch_id + formula_id;
	clause *formula = d_f1 + formula_id * r;
	clause *destination = d_f2 + (branch_id * k + formula_id) * r;
	clause fc = formula[0]; // this might be slow, use __shfl_sync()?

	// check
	if(!(fc.flags & (0x08u << branch_id))) {
		if(lane_id == 0) {
			*valid = 0;
		}

		return;
	}

	for(int i = lane_id; i < r; i += 32) {
		clause cl = formula[i];

		if(c_sat(cl)) { // sprawdzic czy jest dobrze: jak jest nullowalna, to uciekaj
			break;
		}

		for(int l = 0; l < 3; ++l) {
			for(int x = 0; x < branch_id; ++x) {
				if(!(cl.flags & (0x08u << l)) && abs8(cl.l[l]) == abs8(fc.l[x])) {
					cl.flags |= (0x08u + (fc.l[x] < 0)) << l;
				}
			}

			if(cl.l[l] == fc.l[branch_id]) {
				cl.flags |= (0x08u + (fc.l[branch_id] > 0)) << l;
			}
		}

		// check
		if(__any_sync(0xffffffffu, c_inv(cl))) {
			if(lane_id == 0) {
				*valid = 0;
			}

			return;
		}

		destination[i] = cl;
	}
}

/*************************** 1D_SCAN *****************************/

__device__ volatile unsigned int id = 0;
__device__ volatile unsigned int d_p[32];
__device__ volatile unsigned int valid_f;

__inline__ __device__ unsigned int warp_scan(unsigned int v) {
	int lane_id = threadIdx.x & 31;

	for(int i = 1; i < 32; i <<= 1) {
		int _v = __shfl_up_sync(0xffffffffu, v, i);

		if(lane_id >= i) {
			v += abs32(_v);
		}
	}

	return v;
}

__global__ void 1d_scan(unsigned int *d_v, int k, int range_parts, int range) {
	__shared__ partials[33];
	__shared__ prev;
	int tid = blockIdx.x * range + threadIdx.x;
	int warp_id = threadIdx.x >> 5;
	int lane_id = threadIdx.x & 31;

	if(tid == 0) {
		values[0] = 0;
		prev = 0;
	}

	__syncthreads();

	for(int i = 0; i < range_parts && tid < k; tid += 1024) {
		unsigned int v = warp_scan(d_v[tid]);

		if(lane_id == 31) {
			values[warp_id + 1] = v;
		}

		__syncthreads();

		if(warp_id == 0) {
			partials[lane_id] = warp_scan(partials[lane_id]);
		}

		__syncthreads();

		d_v[tid] = v + prev;

		__syncthreads();

		if((tid & 1023) == 1023) {
			prev = abs32(v);
		}
	}

	if((tid & 1023) == 1023) {
		d_p[blockIdx.x] = prev;
		__threadfence();

		if(atomicAdd(&id, 1) == gridDim.x - 1) {
			id = 0;
			d_p[lane_id] = warp_scan(d_p[lane_id]);
		}
	}
}

__global__ void 1d_propagate(unsigned int *d_v, int k, int range_parts, int range) {
	__shared__ int prev;
	int tid = (blockIdx.x + 1) * range + threadIdx.x;

	if(threadIdx.x = 0) {
		prev = d_p[blockIdx.x];
	}

	__syncthreads();

	unsigned int v;

	for(int i = 0; i < range_parts && tid < k; tid += 1024) {
		v = d_v[tid] += prev;
	}

	if(tid == k + 1023) {
		valid_f = v;
	}
}

/************************** 1D_SCATTER ***************************/

__global__ void 1d_scatter(clause *d_f1, clause *d_f2, int *d_v, int k, int r) {
	int warp_id = (blockIdx.x << 5) + (threadIdx.x >> 5);
	unsigned int v = d_v[warp_id];
	unsigned int valid = v & 0x80000000u;
	clause *formula = d_f2 + warp_id * r;
	clause *destination = d_f1 + (valid ? position - 1 : valid_f + warp_id - position) * r;

	for(int i = threadIdx.x & 31; i < r; i += 32) {
		destination[i] = formula[i];
	}
}

/*************************** 2D_SCAN *****************************/

__inline__ __device__ unsigned int warp_scan(unsigned int v, int reminder, int lane_id) {
	for(int i = 1; i < 32; i <<= 1) {
		int _v = __shfl_up_sync(0xffffffffu, v, i);

		if(lane_id >= i && i <= reminder) { // chyba dobrze
			v += abs32(_v);
		}
	}

	return v;
}

__global__ void 2d_scan(clause *d_f1, int *d_v, int k, int r, int range_parts, int range) {
	__shared__ partials[33];
	__shared__ prev;
	int tid = blockIdx.x * range + threadIdx.x;
	int warp_id = threadIdx.x >> 5;
	int lane_id = threadIdx.x & 31;
	int range_start = tid;

	if(tid == 0) {
		values[0] = 0;
		prev = 0;
	}

	__syncthreads();

	for(int i = 0; i < range_parts && tid < k; tid += 1024) {
		int remainder = tid % r; // da sie ifami, ale remainder zwiekszam o 1024%r if(remainder >= r) { remainder -= r; }
	clause cl = d_f1[tid];
	unsigned int satisfied = c_sat(cl) ? 0 : 0x80000001u;
	unsigned int v = warp_scan(satisfied, reminder, lane_id);

	if(lane_id == 31) {
		values[warp_id + 1] = v;
	}

	__syncthreads();

	if(warp_id == 0) {
		partials[lane_id] = warp_scan(partials[lane_id]);
	}

	__syncthreads();

	if(tid - range_start <= reminder) { // chyba dobrze
		d_v[tid] = v + prev;
	}

	__syncthreads();

	if((tid & 1023) == 1023) {
		prev = abs32(v);
	}
	}

	if((tid & 1023) == 1023) {
		d_p[blockIdx.x] = prev;
		__threadfence();

		if(atomicAdd(&id, 1) == gridDim.x - 1) {
			id = 0;
			d_p[lane_id] = warp_scan(d_p[lane_id]);
		}
	}
}

// NIE MA 2D_PROPAGATE

/************************** 2D_SCATTER ***************************/

__global__ void 2d_scatter(clause *d_f1, clause *d_f2, int *d_v, int r) {
	int warp_id = (blockIdx.x << 5) + (threadIdx.x >> 5);
	int shift = warp_id * r;
	int *position = d_v + shift;
	clause *formula = d_f1 + shift;
	clause *destination = d_f2 + shift;

	for(int i = threadIdx.x & 31; i < r; i += 32) {
		unsigned int satisfied = v & 0x80000000u;
		int p = position[i];
		destination[satisfied ? p - 1 : last + warp_id - p] = formula[i];
	}
}

/**************************** SWAP *******************************/

__managed__ bool formula_satisfied = false; // pewnie wyzej umiescic i sprawdzic kiedys???

void swap() {
	
}

/************************** PIPELINE *****************************/
void pipeline(std::vector<clause> &formulas, int n, int r, int s) {
	// in main
}

/************************ EXTRACT_VARS ***************************/

// from a formula, extracts variables

void extract_vars(clause *formula, int r, std::vector<bool> &assignment) {
	for(int i = 0; i < r; ++i) {
		for(int j = 0; j < 3; ++j) {
			int8_t var = formula[i].l[j];
			bool val = formula[i].flags & (0x01u << j);
			bool set = formula[i].flags & (0x08u << j);

			if(set) {
				assignment[abs8(var)] = (var < 0) ^ val; 
			}
		}
	}
}

/**************************** MAIN *******************************/

int main() {
	int n, r, s;
	int number_of_formulas = 1;
	std::vector<clause> formulas(BATCH_SIZE * r);

	clause *d_f1;
	clause *d_f2;
	unsigned int *d_v;
	hipHostMalloc(&d_f1, BATCH_SIZE * r * sizeof(clause));
	hipHostMalloc(&d_f2, BATCH_SIZE * r * sizeof(clause));
	hipHostMalloc(&d_v, BATCH_SIZE * sizeof(unsigned int));

	preprocess<<<0, 0>>>(d_f1, d_f2, d_v, BATCH_SIZE, r);

	while(true) {
		1d_scan<<<0, 0>>>(d_v, number_of_formulas /* ??? */, range_parts, range);
		1d_defrag<<<0, 1024>>>(d_f1);
		2d_scan<<<0, 1024>>>(d_f1, d_v, range_parts, range);
		2d_defrag<<<0, 1024>>>(d_f1, d_f2, d_v, r);
		swap();
		sat_kernel<<<0, 0>>>(d_f1, d_f2, d_v, BATCH_SIZE, r);
	}
}

////////////////////////////////////////////////////////////////////////

struct pair {
	int v;
	int g;
};

__inline__ __device__ pair warp_prefix_scan(int v, int g) {
	int lane_id = threadIdx.x & 31;

	for(int i = 1; i < 32; i *= 2) {
		int _v = __shfl_up_sync(-1, v, i);
		int _g = __shfl_up_sync(-1, g, i);

		if(lane_id >= i) {
			v = g ? v : _v + v;
			g |= _g;
		}
	}

	return {v, g};
}

__device__ int d_v[32];
__device__ int d_g[32];

__global__ void block_prefix_scan(int *d_values, int *d_groups, int n, int parts, int range) {
	__shared__ int values[33];
	__shared__ int groups[33];
	__shared__ int prev_v;
	__shared__ int prev_g;

	if(threadIdx.x == 0) {
		values[0] = 0;
		groups[0] = 0;
		prev_v = 0;
		prev_g = 0;
	}

	__syncthreads();

	int tid = blockIdx.x * range + threadIdx.x;
	int warp_id = threadIdx.x >> 5;
	int lane_id = threadIdx.x & 31;
	int i;

	for(i = 0; i < parts; ++i, tid += 1024) {
		if(tid >= n) {
			return;
		}

		pair p(warp_prefix_scan(d_values[tid], d_groups[tid]));

		if(lane_id == 31) {
			values[warp_id + 1] = p.v;
			groups[warp_id + 1] = p.g;
		}

		__syncthreads();

		if(warp_id == 0) {
			pair q(warp_prefix_scan(values[lane_id], groups[lane_id]));
			values[lane_id] = q.v;
			groups[lane_id] = q.g;
		}

		__syncthreads();

		if(p.g == 0) {
			p.v += values[warp_id];
			p.g |= groups[warp_id];
		}

		if(p.g == 0) {
			p.v += prev_v;
			p.g |= prev_g;
		}

		d_values[tid] = p.v;
		d_groups[tid] = p.g;

		__syncthreads();

		if((tid & 1023) == 1023) {
			prev_v = p.v;
			prev_g = p.g;
		}
	}

	if(i == parts && (tid & 1023) == 1023) {
		d_v[blockIdx.x] = prev_v;
		d_g[blockIdx.x] = prev_g;
	}
}

__global__ void partials() {
	pair p(warp_prefix_scan(d_v[threadIdx.x], d_g[threadIdx.x]));
	d_v[threadIdx.x] = p.v;
	d_g[threadIdx.x] = p.g;
}

__global__ void propagate(int *d_values, int *d_groups, int n, int parts, int range) {
	__shared__ int prev_v;
	int tid = range * (blockIdx.x + 1) + threadIdx.x;

	if(threadIdx.x == 0) {
		prev_v = d_v[blockIdx.x];
	}

	__syncthreads();

	for(int i = 0; i < parts; ++i, tid += 1024) {
		if(tid >= n || d_groups[tid] != 0) {
			return;
		}

		d_values[tid] += prev_v;
	}
}

__global__ void fill_groups(int *d_groups, int *d_groups_original, int m) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid < m - 1) {
		d_groups[d_groups_original[tid] + 1] = 1;

		if(tid == 0) {
			d_groups[0] = 1;
		}
	}
}

void segmentedPrefixSum(std::vector<int>& values, const std::vector<int>& groups) {
	int n = values.size();
	int m = groups.size();
	int parts = (n + 32 * 1024 - 1) / (32 * 1024);
	int range = parts * 1024;
	int blocks = (n + range - 1)/range;
	int *d_values, *d_groups, *d_groups_original;

	hipMalloc(&d_values, n * sizeof(int));
	hipMemcpyAsync(d_values, values.data(), n * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&d_groups_original, m * sizeof(int));
	hipMemcpyAsync(d_groups_original, groups.data(), m * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&d_groups, n * sizeof(int));
	hipMemset(d_groups, 0, n * sizeof(int));

	fill_groups<<<(m + 1023)/1024, 1024>>>(d_groups, d_groups_original, m);
	block_prefix_scan<<<blocks, 1024>>>(d_values, d_groups, n, parts, range);

	if (blocks > 1) {
		partials<<<1, 32>>>();
		propagate<<<blocks - 1, 1024>>>(d_values, d_groups, n, parts, range);
	}

	hipMemcpy(values.data(), d_values, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_values);
	hipFree(d_groups);
	hipFree(d_groups_original);
}
