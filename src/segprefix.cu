#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdint.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define BATCH_SIZE 4096
#define WARPS_NB 10
#define abs8(n) ((n) & 0x7fu)
#define abs32(n) ((n) & 0x7fffffffu)

struct clause {
	/* Field 'flags':
	 *   0x01u - value of literal l[0]
	 *   0x02u - value of literal l[1]
	 *   0x04u - value of literal l[2]
	 *   0x08u - literal l[0] was assigned
	 *   0x10u - literal l[1] was assigned
	 *   0x20u - literal l[2] was assigned
	 * Nullified if: !((c.flags & 0x38u) && (c.flags & 0x07u))
	 * Invalid if: (c.flags & 0x3fu) == 0x2au
	 */
	int8_t l[3];
	uint8_t flags;
};

/************************* PREPROCESS ****************************/

__global__ void preprocess(clause *d_f1, unsigned int *d_v, int r) {
	int warp_id = WARPS_NB * blockIdx.x + (threadIdx.x >> 5); // check
	int lane_id = threadIdx.x & 31;
	clause *formula = d_f1 + warp_id * r;
	unsigned int *valid = d_v + warp_id; // check

	for(int i = threadIdx.x & 31; i < r; ++i) {
		formula[i] = d_f1[i];
	}
	
	int number = warp_id;

	while(number) { // check
		int tmp = number / 3;
		int branch_id = number - 3 * tmp;
		number = tmp;
		clause fc;
		bool fc_found = false;

		for(int i = lane_id; true; i += 32) {
			unsigned int mask1 = __ballot_sync(0xffffffffu, i < r);

			if(i >= r) {
				break;
			}

			clause cl = formula[i];

			if(!fc_found) {
				int has_literals = cl.flags & 0x38u; // ulepszyc pod fc_found i konczenie loop
				int mask2 = __ballot_sync(mask1, has_literals);

				if(!mask2) {
					continue;
				}

				fc_found = true;
				int *ptr_cl = (int *) &cl;
				int src_lane_id = __ffs(mask2) - 1;
				tmp = __shfl_sync(mask1, *ptr_cl, src_lane_id);
				fc = *((clause *) &tmp);
			}

			for(int l = 0; l < 3; ++l) {
				for(int x = 0; x < branch_id; ++x) {
					if(!(cl.flags & (0x08u << l)) && abs8(cl.l[l]) == abs8(fc.l[x])) {
						cl.flags |= (0x08u + (fc.l[x] < 0)) << l;
					}
				}

				if(cl.l[l] == fc.l[branch_id]) {
					cl.flags |= (0x08u + (fc.l[branch_id] > 0)) << l;
				}
			}

			if((cl.flags & 0x3fu) == 0x2au) {
				*valid = 0;
			}

			formula[i] = cl;
		}

		if(!fc_found
	}
}

/************************* SAT_KERNEL ****************************/

/* Triples the number of formulas in a batch and marks invalid/missing ones
 * 
 * d_f - array of formulas and a free space for new formulas
 * d_v - array of flags indicating whether a formula is valid or not
 * k - number of formulas to triple
 * r - total number of clauses
 */
__global__ void sat_kernel(clause *d_f1, clause *d_f2, unsigned int *d_v, int k, int r) {
	int warp_id = WARPS_NB * blockIdx.x + (threadIdx.x >> 5);
	int formula_id = warp_id / 3;
	int branch_id = warp_id - 3 * formula_id;
	unsigned int *valid = d_v + k * branch_id + formula_id;
	clause *formula = d_f1 + formula_id * r;
	clause *destination = d_f2 + (branch_id * k + formula_id) * r;
	clause fc = formula[0]; // this might be slow

	if(!(fc.flags & (0x08u << branch_id))) {
		*valid = 0;
		return;
	}

	for(int i = threadIdx.x & 31; i < r; i += 32) {
		clause cl = formula[i]; // dodac check czy formula jest nullowalna

		for(int l = 0; l < 3; ++l) {
			for(int x = 0; x < branch_id; ++x) {
				if(!(cl.flags & (0x08u << l)) && abs8(cl.l[l]) == abs8(fc.l[x])) {
					cl.flags |= (0x08u + (fc.l[x] < 0)) << l;
				}
			}

			if(cl.l[l] == fc.l[branch_id]) {
				cl.flags |= (0x08u + (fc.l[branch_id] > 0)) << l;
			}
		}

		if((cl.flags & 0x3fu) == 0x2au) {
			*valid = 0;
		}

		destination[i] = cl;
	}
}

/*************************** 1D_SCAN *****************************/

__device__ volatile unsigned int id = 0;
__device__ volatile unsigned int d_p[32];
__device__ volatile unsigned int last;

__inline__ __device__ unsigned int warp_scan(unsigned int v) {
	int lane_id = threadIdx.x & 31;

	for(int i = 1; i < 32; i <<= 1) {
		int _v = __shfl_up_sync(0xffffffffu, v, i);

		if(lane_id >= i) {
			v += abs32(_v);
		}
	}

	return v;
}

__global__ void 1d_scan(unsigned int *d_v, int k, int range_parts, int range) {
	__shared__ partials[33];
	__shared__ prev;
	int tid = blockIdx.x * range + threadIdx.x;
	int warp_id = threadIdx.x >> 5;
	int lane_id = threadIdx.x & 31;

	if(tid == 0) {
		values[0] = 0;
		prev = 0;
	}

	__syncthreads();

	for(int i = 0; i < range_parts && tid < k; tid += 1024) {
		unsigned int v = warp_scan(d_v[tid]);

		if(lane_id == 31) {
			values[warp_id + 1] = v;
		}

		__syncthreads();

		if(warp_id == 0) {
			partials[lane_id] = warp_scan(partials[lane_id]);
		}

		__syncthreads();

		d_v[tid] = v + prev;

		__syncthreads();

		if((tid & 1023) == 1023) {
			prev = abs32(v);
		}
	}

	if((tid & 1023) == 1023) {
		d_p[blockIdx.x] = prev;
		__threadfence();

		if(atomicAdd(&id, 1) == gridDim.x - 1) {
			id = 0;
			d_p[lane_id] = warp_scan(d_p[lane_id]);
		}
	}
}

__global__ void 1d_propagate(unsigned int *d_v, int k, int range_parts, int range) {
	__shared__ int prev;
	int tid = (blockIdx.x + 1) * range + threadIdx.x;

	if(threadIdx.x = 0) {
		prev = d_p[blockIdx.x];
	}

	__syncthreads();

	unsigned int v;

	for(int i = 0; i < range_parts && tid < k; tid += 1024) {
		v = d_v[tid] += prev;
	}

	if(tid == k + 1023) {
		last = v;
	}
}

/************************** 1D_SCATTER ***************************/

__global__ void 1d_scatter(clause *d_f1, clause *d_f2, int *d_v, int k, int r) {
	int warp_id = (blockIdx.x << 5) + (threadIdx.x >> 5);
	unsigned int v = d_v[warp_id];
	unsigned int valid = v & 0x80000000u;
	clause *formula = d_f2 + warp_id * r;
	clause *destination = d_f1 + (valid ? position - 1 : last + warp_id - position) * r;

	for(int i = threadIdx.x & 31; i < r; i += 32) {
		destination[i] = formula[i];
	}
}

/*************************** 2D_SCAN *****************************/

__inline__ __device__ unsigned int warp_scan(unsigned int v, int reminder, int lane_id) {
	for(int i = 1; i < 32; i <<= 1) {
		int _v = __shfl_up_sync(0xffffffffu, v, i);

		if(lane_id >= i && i <= reminder) { // chyba dobrze
			v += abs32(_v);
		}
	}

	return v;
}

__global__ void 2d_scan(clause *d_f1, int *d_v, int k, int r, int range_parts, int range) {
	__shared__ partials[33];
	__shared__ prev;
	int tid = blockIdx.x * range + threadIdx.x;
	int warp_id = threadIdx.x >> 5;
	int lane_id = threadIdx.x & 31;
	int range_start = tid;

	if(tid == 0) {
		values[0] = 0;
		prev = 0;
	}

	__syncthreads();

	for(int i = 0; i < range_parts && tid < k; tid += 1024) {
		int remainder = tid % r; // da sie ifami, ale remainder zwiekszam o 1024%r if(remainder >= r) { remainder -= r; }
	clause cl = d_f1[tid];
	unsigned int satisfied = (cl.flags & 0x07u) ? 0 : 0x80000001u;
	unsigned int v = warp_scan(satisfied, reminder, lane_id);

	if(lane_id == 31) {
		values[warp_id + 1] = v;
	}

	__syncthreads();

	if(warp_id == 0) {
		partials[lane_id] = warp_scan(partials[lane_id]);
	}

	__syncthreads();

	if(tid - range_start <= reminder) { // chyba dobrze
		d_v[tid] = v + prev;
	}

	__syncthreads();

	if((tid & 1023) == 1023) {
		prev = abs32(v);
	}
	}

	if((tid & 1023) == 1023) {
		d_p[blockIdx.x] = prev;
		__threadfence();

		if(atomicAdd(&id, 1) == gridDim.x - 1) {
			id = 0;
			d_p[lane_id] = warp_scan(d_p[lane_id]);
		}
	}
}

// NIE MA 2D_PROPAGATE

/************************** 2D_SCATTER ***************************/

__global__ void 2d_scatter(clause *d_f1, clause *d_f2, int *d_v, int r) {
	int warp_id = (blockIdx.x << 5) + (threadIdx.x >> 5);
	int shift = warp_id * r;
	int *position = d_v + shift;
	clause *formula = d_f1 + shift;
	clause *destination = d_f2 + shift;

	for(int i = threadIdx.x & 31; i < r; i += 32) {
		unsigned int satisfied = v & 0x80000000u;
		int p = position[i];
		destination[satisfied ? p - 1 : last + warp_id - p] = formula[i];
	}
}

/**************************** SWAP *******************************/

void swap() {

}

/************************** PIPELINE *****************************/
void pipeline() {
	while(true) {
		1d_scan<<<0, 0>>>();
		1d_defrag<<<0, 1024>>>();
		2d_scan<<<0, 1024>>>();
		2d_defrag<<<0, 1024>>>();
		swap();
		sat_kernel<<<0, 0>>>();
	}
}

/**************************** MAIN *******************************/

int main() {
	int n;
	int r;
	int s;
	unsigned int *d_f1;
	unsigned int *d_f2;
	hipHostMalloc(&d_f_1, ... * r * sizeof(clause));
	hipHostMalloc(&d_f_2, ... * r * sizeof(clause));
}

////////////////////////////////////////////////////////////////////////

struct pair {
	int v;
	int g;
};

__inline__ __device__ pair warp_prefix_scan(int v, int g) {
	int lane_id = threadIdx.x & 31;

	for(int i = 1; i < 32; i *= 2) {
		int _v = __shfl_up_sync(-1, v, i);
		int _g = __shfl_up_sync(-1, g, i);

		if(lane_id >= i) {
			v = g ? v : _v + v;
			g |= _g;
		}
	}

	return {v, g};
}

__device__ int d_v[32];
__device__ int d_g[32];

__global__ void block_prefix_scan(int *d_values, int *d_groups, int n, int parts, int range) {
	__shared__ int values[33];
	__shared__ int groups[33];
	__shared__ int prev_v;
	__shared__ int prev_g;

	if(threadIdx.x == 0) {
		values[0] = 0;
		groups[0] = 0;
		prev_v = 0;
		prev_g = 0;
	}

	__syncthreads();

	int tid = blockIdx.x * range + threadIdx.x;
	int warp_id = threadIdx.x >> 5;
	int lane_id = threadIdx.x & 31;
	int i;

	for(i = 0; i < parts; ++i, tid += 1024) {
		if(tid >= n) {
			return;
		}

		pair p(warp_prefix_scan(d_values[tid], d_groups[tid]));

		if(lane_id == 31) {
			values[warp_id + 1] = p.v;
			groups[warp_id + 1] = p.g;
		}

		__syncthreads();

		if(warp_id == 0) {
			pair q(warp_prefix_scan(values[lane_id], groups[lane_id]));
			values[lane_id] = q.v;
			groups[lane_id] = q.g;
		}

		__syncthreads();

		if(p.g == 0) {
			p.v += values[warp_id];
			p.g |= groups[warp_id];
		}

		if(p.g == 0) {
			p.v += prev_v;
			p.g |= prev_g;
		}

		d_values[tid] = p.v;
		d_groups[tid] = p.g;

		__syncthreads();

		if((tid & 1023) == 1023) {
			prev_v = p.v;
			prev_g = p.g;
		}
	}

	if(i == parts && (tid & 1023) == 1023) {
		d_v[blockIdx.x] = prev_v;
		d_g[blockIdx.x] = prev_g;
	}
}

__global__ void partials() {
	pair p(warp_prefix_scan(d_v[threadIdx.x], d_g[threadIdx.x]));
	d_v[threadIdx.x] = p.v;
	d_g[threadIdx.x] = p.g;
}

__global__ void propagate(int *d_values, int *d_groups, int n, int parts, int range) {
	__shared__ int prev_v;
	int tid = range * (blockIdx.x + 1) + threadIdx.x;

	if(threadIdx.x == 0) {
		prev_v = d_v[blockIdx.x];
	}

	__syncthreads();

	for(int i = 0; i < parts; ++i, tid += 1024) {
		if(tid >= n || d_groups[tid] != 0) {
			return;
		}

		d_values[tid] += prev_v;
	}
}

__global__ void fill_groups(int *d_groups, int *d_groups_original, int m) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid < m - 1) {
		d_groups[d_groups_original[tid] + 1] = 1;

		if(tid == 0) {
			d_groups[0] = 1;
		}
	}
}

void segmentedPrefixSum(std::vector<int>& values, const std::vector<int>& groups) {
	int n = values.size();
	int m = groups.size();
	int parts = (n + 32 * 1024 - 1) / (32 * 1024);
	int range = parts * 1024;
	int blocks = (n + range - 1)/range;
	int *d_values, *d_groups, *d_groups_original;

	hipMalloc(&d_values, n * sizeof(int));
	hipMemcpyAsync(d_values, values.data(), n * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&d_groups_original, m * sizeof(int));
	hipMemcpyAsync(d_groups_original, groups.data(), m * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&d_groups, n * sizeof(int));
	hipMemset(d_groups, 0, n * sizeof(int));

	fill_groups<<<(m + 1023)/1024, 1024>>>(d_groups, d_groups_original, m);
	block_prefix_scan<<<blocks, 1024>>>(d_values, d_groups, n, parts, range);

	if (blocks > 1) {
		partials<<<1, 32>>>();
		propagate<<<blocks - 1, 1024>>>(d_values, d_groups, n, parts, range);
	}

	hipMemcpy(values.data(), d_values, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_values);
	hipFree(d_groups);
	hipFree(d_groups_original);
}
